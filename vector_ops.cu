/* File: mpi_vector_ops.c
 * COMP 137-1 Spring 2020
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
char* infile = NULL;
char* outfile = NULL;

int readInputFile(char* filename, long* n_p, double* x_p, double** A_p, double** B_p)
{
    long i;
    FILE* fp = fopen(filename, "r");
    if (fp == NULL) return 0;
    fscanf(fp, "%ld\n", n_p);
    fscanf(fp, "%lf\n", x_p);
    *A_p = (double *)malloc(*n_p*sizeof(double));
    *B_p = (double *)malloc(*n_p*sizeof(double));
    for (i=0; i<*n_p; i++) fscanf(fp, "%lf\n", (*A_p)+i);
    for (i=0; i<*n_p; i++) fscanf(fp, "%lf\n", (*B_p)+i);
    return 1;
}

__global__ void cudaSolution(double number, double* W, double* X, double* Y, double* Z) {

        Y[threadIdx.x] = number*W[threadIdx.x];
        Z[threadIdx.x] = W[threadIdx.x]*X[threadIdx.x];
        
}        


int writeOutputFile(char* filename, long n,  double* C, double* D)
{
    long i;
    FILE* fp = fopen(filename, "w");
    if (fp == NULL) return 0;
    fprintf(fp, "%ld\n", n);
    for (i=0; i<n; i++) fprintf(fp, "%lf\n", C[i]);
    for (i=0; i<n; i++) fprintf(fp, "%lf\n", D[i]);
    return 1;
}



int main(int argc, char* argv[])
{
    long    n=0;     /* size of input arrays */
    double  x;       /* input scalar */
    double* A;       /* input vector */
    double* B;       /* input vector */
    double* C;       /* output vector xA */
    double* D;       /* output vector A*B */
    double* E_D;
    double* F_D;
    double* G_D; 
    double* H_D; 
    

    /* read input data */
    if (argc<3)
    {
        n = -1;
        fprintf(stderr, "Command line arguments are required.\n");
        fprintf(stderr, "argv[1] = name of input file\n");
        fprintf(stderr, "argv[2] = name of input file\n");
    }
    else
    {
        infile = argv[1];
        outfile = argv[2];
        if (!readInputFile(infile, &n, &x, &A, &B))
        {
            fprintf(stderr, "Error opening input files. Aborting.\n");
            n = -1;
        }
    }
    
    double sizesGrid = 1;
    double sizesBlock = 1024;

       if (n > sizesBlock) {
         sizesGrid = (int)ceil((float) n/sizesBlock);
       }
         int sizing = n * sizeof(double);

       hipMalloc(&E_D, sizing);
       hipMalloc(&F_D, sizing);
       hipMalloc(&G_D, sizing);
       hipMalloc(&H_D, sizing);

       hipMemcpy(E_D, A, sizing, hipMemcpyHostToDevice);
       hipMemcpy(F_D, B, sizing, hipMemcpyHostToDevice);

       cudaSolution<<<sizesGrid,sizesBlock>>>(x, E_D, F_D, G_D, H_D); 
       C = (double *)malloc(sizing * sizeof(double));
       D = (double *)malloc(sizing * sizeof(double));

      hipMemcpy(C, G_D, sizing, hipMemcpyDeviceToHost);
      hipMemcpy(D, H_D, sizing, hipMemcpyDeviceToHost);

    if (n < 0)
    {
        fprintf(stderr, "Aborting task due to input errors.\n");
        exit(1);
    }

    hipDeviceSynchronize();

    if (!writeOutputFile(outfile, n, C, D))
    {
        fprintf(stderr, "Error opening output file. Aborting.\n");
        exit(1);
    }
    
/* free all dynamic memory allocation */
    free(A);
    free(B);
    free(C);
    free(D);
    hipFree(E_D);
    hipFree(F_D);
    hipFree(G_D);
    hipFree(H_D);

    hipDeviceSynchronize();
 
    return 0;
}
